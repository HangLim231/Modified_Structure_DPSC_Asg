// File: cuda/conv2d_kernel.cu

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <cmath>
using namespace std;

// Kernel for 2D convolution forward pass
__global__ void conv2d_forward_kernel(
    const float* input, // input feature map (N x C_in x H x W)
    const float* weights, // Convolutional Filter (C_out x C_in x K x K)
    const float* bias, // Bias Terms (C_out)
    float* output, // Output feature map (N x C_out x H_out x W_out)
    int N, int C_in, int H, int W, // Input dimensions batch size, input_channels, height, width, output_channels, kernel size
    int C_out, int K)
{
    int n = blockIdx.z; // Batch dimension
    int co = blockIdx.y; // Output channel dimension
    int hw = blockIdx.x * blockDim.x + threadIdx.x; //Spatial dimension

    int H_out = H - K + 1;
    int W_out = W - K + 1;
    if (hw >= H_out * W_out) return;

    int h = hw / W_out;
    int w = hw % W_out;

    float sum = bias[co];

    // Convolution operation
    for (int ci = 0; ci < C_in; ci++) {
        for (int p = 0; p < K; p++) {
            for (int q = 0; q < K; q++) {
                int in_h = h + p;
                int in_w = w + q;
                int input_idx = n * (C_in * H * W) + ci * (H * W) + in_h * W + in_w;
                int weight_idx = co * (C_in * K * K) + ci * (K * K) + p * K + q;
                sum += input[input_idx] * weights[weight_idx];
            }
        }
    }

    int out_idx = n * (C_out * H_out * W_out) + co * (H_out * W_out) + h * W_out + w;
    output[out_idx] = fmaxf(0.0f, sum);
}

// Kernel for 2D convolution forward pass
void conv2d_forward(
    const float* d_input,
    const float* d_weights,
    const float* d_bias,
    float* d_output,
    int N, int C_in, int H, int W,
    int C_out, int K)
{
    int H_out = H - K + 1;
    int W_out = W - K + 1;
    dim3 grid((H_out * W_out + 255) / 256, C_out, N);
    dim3 block(256);

    conv2d_forward_kernel <<<grid, block>>> (
        d_input, d_weights, d_bias, d_output,
        N, C_in, H, W, C_out, K
        );
    hipDeviceSynchronize();
}
